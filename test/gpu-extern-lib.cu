
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void inc_kernel(int32_t *in, int32_t width, int32_t height, int32_t v, int32_t *out) {
    int gx = threadIdx.x + blockDim.x * blockIdx.x;
    int gy = threadIdx.y + blockDim.y * blockIdx.y;

    if (gx < width && gy < height) {
        out[gy * width + gx] = in[gy * width + gx] + v;
    }
}

extern "C"
void call_inc_kernel(int32_t *in, int32_t width, int32_t height, int32_t v, int32_t *out) {
    dim3 block_size(16, 16);
    int bx = (width + block_size.x - 1)/block_size.x;
    int by = (height + block_size.y - 1)/block_size.y;
    dim3 grid_size(bx, by);

    inc_kernel<<<grid_size, block_size>>>(in, width, height, v, out);
}

